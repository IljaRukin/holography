#include "hip/hip_runtime.h"
//#include "mex.h"

#include "hip/hip_runtime.h"
#include ""
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include <ctime>
#include <time.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <hipfft/hipfft.h>
#include <fstream>

__global__ void main_func(float2 * signal,
                        const unsigned int M,
                        const unsigned int N)
{
	// CUFFT plan
	hipfftHandle plan;
	hipfftPlan2d(&plan, M, N, HIPFFT_C2C);

	// FFT
	hipfftExecC2C(plan, (hipfftComplex *)signal, (hipfftComplex *)signal, HIPFFT_FORWARD); //HIPFFT_BACKWARD

}