
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
//stored in: C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v11.1\include\cuComplex.h

//determine block & thread, calculate unique index
__device__ size_t calculateGlobalIndex() {
    size_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
    size_t const localThreadIdx = threadIdx.x + blockDim.x * threadIdx.y;
    size_t const threadsPerBlock = blockDim.x*blockDim.y;
    return localThreadIdx + globalBlockIndex*threadsPerBlock;
}

__global__ void main_func( 
                      const float2 * wavefield,
                      float2 * out,
                      const unsigned int M ,
                      const unsigned int N ) {
    // current thread
    size_t const numel = M*N*M*N;
    int const idx = numel-1 - calculateGlobalIndex(); //optimize: reversed direction
    
    if (idx < 0){
        return;
    }

    // coordinates
    // W(y,x,fy,fx)={{{{fx},{fx}},{{fx},{fx}}},{{{fx},{fx}},{{fx},{fx}}}}
    //                | 0 fy  1 |,| 0 fy  1 | , | 0 fy  1 |,| 0 fy  1 |
    //               |     0     x     1     |,|     0     x     1     |
    //              |            0            y            1            |
    int fx = idx/(N*M*N);
    int fy = idx/(M*N) - fx*(N);
    int x = idx/(N) - fx*(N*M) - fy*(M);
    int y = idx - fx*(N*M*N) - fy*(M*N) - x*(N);
//---
//     int fx = idx/(M*N);          = idx/(N*M*N);
//     int fy = idx/(M*N) - fx;     = idx/(M*N) - fx*(N);
//     int x = idx - fx*(M);        = idx/(N) - fx*(N*M) - fy*(M);
//     int y = idx - fx*(M) - x;    = idx - fx*(N*M*N) - fy*(M*N) - x*(N);
//---
    int xa = x +(fx-(float)(M/2));
    int xb = x -(fx-(float)(M/2));
    int ya = y +(fy-(float)(N/2));
    int yb = y -(fy-(float)(N/2));

    //W[idx]->W(y,x,fy,fx)
    //W[idx2]->W(fx,fy,x,y)
    int idx2 = fx+M*fy+N*M*x+M*N*M*y;
    if (idx2 >= numel){
        return;
    }

    //shift & multiply
    if ( ( xa > -1 ) && ( xb > -1 ) && ( ya > -1 ) && ( yb > -1 ) && ( xa < M ) && ( xb < M ) && ( ya < N ) && ( yb < N ) ) {
         out[idx2] = hipCmulf( wavefield[xa+ya*M] , hipConjf(wavefield[xb+yb*M]) );
    }

    //test
//     out[idx2] = make_cuFloatComplex((float)x,(float)y);
//     out[idx2] = make_cuFloatComplex((float)fx,(float)fy);
}