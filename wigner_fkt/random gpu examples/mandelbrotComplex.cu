

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
//stored in: C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v11.1\include\cuComplex.h
//#include <thrust/complex.h>
//thrust::complex<float>

//determine block & thread, calculate unique index
__device__ size_t calculateGlobalIndex() {
    size_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
    size_t const localThreadIdx = threadIdx.x + blockDim.x * threadIdx.y;
    size_t const threadsPerBlock = blockDim.x*blockDim.y;
    return localThreadIdx + globalBlockIndex*threadsPerBlock;
}

//x^2
__host__ __device__ static __inline__ float squaref (float x)
{
    float square;
    square = x*x;
    return square;
}

//z^2
__host__ __device__ static __inline__ hipFloatComplex cuCsquaref (hipFloatComplex z)
{
    hipFloatComplex square;
    square = make_hipFloatComplex( squaref(hipCrealf(z)) - squaref(hipCimagf(z)), 2*hipCrealf(z)*hipCimagf(z) );
    return square;
}

//|z|^2
__host__ __device__ static __inline__ float cuCabs2f (hipFloatComplex z)
{
    float abs2;
    abs2 = squaref(hipCrealf(z)) + squaref(hipCimagf(z));
    return abs2;
}

// calculate mandelbrot set
__device__ unsigned int doIterations( const float2 z0,
                                      unsigned int const maxIters ) {
    float2 z = z0;
    unsigned int counti = 1;

    // Loop until escape
    //while ( ( counti <= maxIters ) && (cuCrealf( cuCmulf( cuConjf(z) , z ) ) <= 4.0) ) {
    while ( ( counti <= maxIters ) && (cuCabs2f(z) <= 4.0) ) {
        ++counti;
        //z = cuCaddf( cuCmulf( z , z ) , z0 );
        z = hipCaddf( cuCsquaref( z ) , z0 );
    }
    return counti;
}

// preprocess input
//float2 is identical to cuFloatComplex
__global__ void main_func( 
                      unsigned int * out, 
                      const float2 * z,
                      const unsigned int maxIters,
                      const unsigned int numel ) {
    // current thread
    size_t const idx = calculateGlobalIndex();

    // quit on thread overflow
    if (idx >= numel) {
        return;
    }

    // compute madelbrot set value
    out[idx] = doIterations( z[idx], maxIters );
}