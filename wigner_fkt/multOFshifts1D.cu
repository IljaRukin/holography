
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
//stored in: C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v11.1\include\cuComplex.h

//determine block & thread, calculate unique index
__device__ size_t calculateGlobalIndex() {
    size_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
    size_t const localThreadIdx = threadIdx.x + blockDim.x * threadIdx.y;
    size_t const threadsPerBlock = blockDim.x*blockDim.y;
    return localThreadIdx + globalBlockIndex*threadsPerBlock;
}

__global__ void main_func( 
                      const float2 * wavefield,
                      float2 * out,
                      const unsigned int M ) {
    // current thread
    size_t const numel = M*M;
    int const idx = numel-1 - calculateGlobalIndex(); //optimize: reversed direction
    
    if (idx < 0){
        return;
    }

    // coordinates
    // W(fx,x)={{x} , {x}}
    //         | 0 fx  1 |
    int fx = idx/M; // [0,...0,1,...1,M-1,...M-1]
    int x = idx - fx*M; // [0,1,...,M-1,0,1,...,M-1,...]
    int xa = x +(fx-((float)M)/2);
    int xb = x -(fx-((float)M)/2);

    int idx2 = fx+M*x;
    if (idx2 >= numel){
        return;
    }
    //W[idx]->W(x,fx)
    //W[idx2]->W(fx,x)

    //shift & multiply
    if ( ( xa > -1 ) && ( xb > -1 ) && ( xa < M ) && ( xb < M ) ) {
        out[idx2] = hipCmulf( wavefield[xa] , hipConjf(wavefield[xb]) );
    }

    //test
//     out[idx] = make_cuFloatComplex((float)x,(float)fx);
//     out[idx].x = x; out[idx].y = fx;
//     out[idx] = make_cuFloatComplex((float)x,(float)(fx-((float)M)/2));
//     out[idx2] = make_cuFloatComplex((float)x,(float)fx);
//     out[idx2] = make_cuFloatComplex((float)x,(float)(fx-((float)M)/2));
}