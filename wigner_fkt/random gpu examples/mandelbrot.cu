
#include <hip/hip_runtime.h>

//determine block & thread, calculate unique index
__device__ size_t calculateGlobalIndex() {
    size_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
    size_t const localThreadIdx = threadIdx.x + blockDim.x * threadIdx.y;
    size_t const threadsPerBlock = blockDim.x*blockDim.y;
    return localThreadIdx + globalBlockIndex*threadsPerBlock;
}

// calculate mandelbrot set
__device__ unsigned int doIterations( float const x0,
                                      float const y0,
                                      unsigned int const maxIters ) {
    float xi = x0;
    float oldxi;
    float yi = y0;
    unsigned int counti = 1;

    // Loop until escape
    while ( ( counti <= maxIters ) && ((xi*xi + yi*yi) <= 4.0) ) {
        ++counti;
        oldxi = xi;
        // real part
        xi = xi*xi - yi*yi + x0;
        // imaginary part
        yi = 2*oldxi*yi + y0;
    }
    return counti;
}

// preprocess input
__global__ void main_func( 
                      unsigned int * out, 
                      const float * x, 
                      const float * y,
                      const unsigned int maxIters,
                      const unsigned int numel ) {
    // current thread
    size_t const idx = calculateGlobalIndex();

    // quit on thread overflow
    if (idx >= numel) {
        return;
    }
    
    // x,y coordinates
    float x0 = x[idx];
    float y0 = y[idx];

    // compute madelbrot set value
    unsigned int const count = doIterations( x0, y0, maxIters );
    out[idx] = count;
}